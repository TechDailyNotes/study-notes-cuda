#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void whoami(void) {
    int block_id =
        blockIdx.x +
        blockIdx.y * gridDim.x +
        blockIdx.z * gridDim.x * gridDim.y;

    int block_offset =
        block_id *
        blockDim.x * blockDim.y * blockDim.z;

    int thread_local_id =
        threadIdx.x +  
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int thread_global_id = block_offset + thread_local_id;

    printf(
        "%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        thread_global_id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_local_id
    );
    // printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(int argc, char **argv) {
    // Step 1: Define dimensions of block and grid.
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;
    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z); // 3d cube of shape 2*3*4 = 24
    dim3 threadsPerBlock(t_x, t_y, t_z); // 3d cube of shape 4*4*4 = 64

    // Step 2: Call device kernel.
    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}
